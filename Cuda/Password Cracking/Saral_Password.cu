#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
/****************************************************************************
  This program gives an example of a poor way to implement a password cracker
  in CUDA C. It is poor because it acheives this with just one thread, which
  is obviously not good given the scale of parallelism available to CUDA
  programs.
  
  The intentions of this program are:
    1) Demonstrate the use of __device__ and __global__ functions
    2) Enable a simulation of password cracking in the absence of library 
       with equivalent functionality to libcrypt. The password to be found
       is hardcoded into a function called is_a_match.   

  Compile and run with:
    nvcc -o Saral_Password Saral_Password.cu
    ./Saral_Password
   
  Dr Kevan Buckley, University of Wolverhampton, 2018
*****************************************************************************/

/****************************************************************************
  This function returns 1 if the attempt at cracking the password is 
  identical to the plain text password string stored in the program. 
  Otherwise,it returns 0.
*****************************************************************************/

__device__ int is_a_match(char *attempt) {
	char mypassword1[] = "SA9780";
	char mypassword2[] = "AS3145";
	char mypassword3[] = "EE4652";
	char mypassword4[] = "BB2565";


	char *p = attempt;
	char *r = attempt;
	char *a = attempt;
	char *t = attempt;
	char *p1 = mypassword1;
	char *p2 = mypassword2;
	char *p3 = mypassword3;
	char *p4 = mypassword4;

	while(*p == *p1) { 
		if(*p == '\0') 
		{
			printf("Password: %s\n",mypassword1);
			break;
		}

		p++;
		p1++;
	}
	
	while(*r == *p2) { 
		if(*r == '\0') 
		{
			printf("Password: %s\n",mypassword2);
			break;
		}

		r++;
		p2++;
	}

	while(*a == *p3) { 
		if(*a == '\0') 
		{
			printf("Password: %s\n",mypassword3);
			break;
		}

		a++;
		p3++;
	}

	while(*t == *p4) { 
		if(*t == '\0') 
		{
			printf("Password: %s\n",mypassword4);
			return 1;
		}

		t++;
		p4++;
	}
	return 0;

}

__global__ void  kernel() {
	char s1,s2,s3,s4;

	char password[7];
	password[6] = '\0';

	int i = blockIdx.x+65;
	int j = threadIdx.x+65;
	char firstMatch = i; 
	char secondMatch = j; 

	password[0] = firstMatch;
	password[1] = secondMatch;
	for(s1='0'; s1<='9'; s1++){
		for(s2='0'; s2<='9'; s2++){
			for(s3='0'; s3<='9'; s3++){
				for(s4='0'; s4<='9'; s4++){
					password[2] = s1;
					password[3] = s2;
					password[4] = s3;
					password[5] = s4; 
					if(is_a_match(password)) {
					} 
					else {
	     			//printf("tried: %s\n", password);		  
					}
				}
			}
		}
	}
}

int time_difference(struct timespec *start, 
	struct timespec *finish, 
	long long int *difference) {
	long long int ds =  finish->tv_sec - start->tv_sec; 
	long long int dn =  finish->tv_nsec - start->tv_nsec; 
	if(dn < 0 ) {
		ds--;
		dn += 1000000000; 
	} 
	*difference = ds * 1000000000 + dn;
	return !(*difference > 0);
}


int main() {

	struct  timespec start, finish;
	long long int time_elapsed;
	clock_gettime(CLOCK_MONOTONIC, &start);

	kernel <<<26,26>>>();
	hipDeviceSynchronize();

	clock_gettime(CLOCK_MONOTONIC, &finish);
	time_difference(&start, &finish, &time_elapsed);
	printf("Time elapsed was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed/1.0e9)); 

	return 0;
}


